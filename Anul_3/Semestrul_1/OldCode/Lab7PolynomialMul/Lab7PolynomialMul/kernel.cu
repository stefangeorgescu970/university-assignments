#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <time.h>
#include <stdlib.h>

#define MAX(a, b) (a) >= (b) ? (a) : (b)
#define MIN(a, b) (a) <= (b) ? (a) : (b)


void printPoly(int * polyCoefficients, int size)
{
	for (int i = 0; i < size; i++) {
		std::cout << (polyCoefficients[i] >= 0 ? "+ " : "- ");
		std::cout << std::abs(polyCoefficients[i]);
		std::cout << " * X^";
		std::cout << i;
		std::cout << " ";
	}
}

hipError_t simpleMulWithCuda(int *result, const int *a, const int *b, unsigned int sizeA, unsigned int sizeB, unsigned int numberOfThreads);
hipError_t karatsubaMulWithCuda(int *result, const int *a, const int *b, unsigned int sizeA, unsigned int sizeB);

__global__ void simpleMulKernel(int *result, const int *a, const int *b, const int sizeA, const int sizeB, const int numberOfThreads)
{
	int sizeResult = sizeA + sizeB - 1;
	int startPosition = (threadIdx.x * sizeResult) / numberOfThreads;
	int endPosition = ((threadIdx.x + 1) * sizeResult) / numberOfThreads;
	for (int position = startPosition; position < endPosition; position++) {
		int sum = 0;
		int begin = (position < sizeB ? 0 : position - sizeB + 1);
		int end = sizeA <= position + 1 ? sizeA : position + 1;
		for (int i = begin; i < end; i++) {
			sum += a[i] * b[position - i];
		}
		result[position] = sum;
	}
}

__global__ void copyArrayOfRange(int* destination, const int* source, const int begin, const int end)
{
	for (int i = begin; i < end; i++) 
	{
		destination[i - begin] = source[i];
	}
}

__global__ void polyAdd(int* result, const int* a, const int* b, const int sizeA, const int sizeB)
{
	int limit = MAX(sizeA, sizeB);//sizeA >= sizeB ? sizeA : sizeB;
	if (sizeA == limit) 
	{
		while (limit > sizeB) 
		{
			result[limit - 1] = a[limit - 1];
			limit--;
		}
	}
	else 
	{
		while (limit > sizeA) 
		{
			result[limit - 1] = b[limit - 1];
			limit--;
		}
	}
	for (int i = 0; i < limit; i++) 
	{
		result[i] = a[i] + b[i];
	}
}

__global__ void polyShiftKernel(int* result, const int* source, const int size, const int shiftPositions)
{
	int i;
	for (i = 0; i < shiftPositions; i++)
	{
		result[i] = 0;
	}
	for (i = 0; i < size; i++)
	{
		result[i + shiftPositions] = source[i];
	}
}

__global__ void polySubKernel(int* result, const int* a, const int* b, const int sizeA, const int sizeB)
{
	int limit = MAX(sizeA, sizeB);// sizeA >= sizeB ? sizeA : sizeB;
	if (sizeA == sizeB) 
	{
		while (limit > 1 && a[limit - 1] == b[limit - 1]) 
		{
			result[limit - 1] = 0;
			limit--;
		}
	}
	else 
	{
		if (sizeA == limit) 
		{
			while (limit > sizeB) 
			{
				result[limit - 1] = a[limit - 1];
				limit--;
			}
		}
		else 
		{
			while (limit > sizeA) 
			{
				result[limit - 1] = -b[limit - 1];
				limit--;
			}
		}
	}
	for (int i = limit - 1; i >= 0; i--) 
	{
		result[i] = a[i] - b[i];
	}
}

__global__ void karatsubaMulKernel(int *result, const int *a, const int *b, const int sizeA, const int sizeB)
{
	//karatsubaMulKernel<<<1, 1>>>(result, a, b, sizeA, sizeB, numberOfThreads - 1);
	hipError_t cudaStatus;// = hipDeviceSynchronize();
	//if (cudaStatus != hipSuccess) {
		//fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	//}
	//printf("ENTRY sizes: %d %d\n", sizeA, sizeB);

	//printf("%d %d pos 1\n", a[0], b[0]);

	/*if (a[0] == 2 && a[1] == 3) {
		printf("A: ");
		for (int i = 0; i < sizeA; i++)
		{
			printf("%d ", a[i]);
		}
		printf("\n");
		printf("B: ");
		for (int i = 0; i < sizeB; i++)
		{
			printf("%d ", b[i]);
		}
		printf("\n");
		printf("\n");
	}*/


	//printf("%d %d pos 2\n", a[0], b[0]);
	if (sizeA < 2 || sizeB < 2) {
		simpleMulKernel <<<1, 1 >>>(result, a, b, sizeA, sizeB, 1);
		hipDeviceSynchronize();
		/*printf("RESULT: ");
		for (int i = 0; i < (sizeA + sizeB - 1); i++)
		{
			printf("%d ", result[i]);
		}
		printf("\n");
		printf("\n");*/
		return;
	}
	int m = sizeA > sizeB ? sizeA : sizeB;
	const int halfPoint = m / 2;
	const int sizeH = sizeA;

	int* highA;
	hipMalloc((void**)&highA, (sizeA - halfPoint) * sizeof(int));
	int* lowA;
	hipMalloc((void**)&lowA, halfPoint * sizeof(int));
	int* highB;
	hipMalloc((void**)&highB, (sizeB - halfPoint) * sizeof(int));
	int* lowB;
	hipMalloc((void**)&lowB, halfPoint * sizeof(int));

	//printf("%d %d sizes hA %d lA %d hB %d lb %d\n", a[0], b[0], (sizeA - halfPoint), halfPoint, (sizeB - halfPoint), halfPoint);

	copyArrayOfRange <<<1, 1 >>>(highA, a, halfPoint, sizeA);
	copyArrayOfRange <<<1, 1 >>>(lowA, a, 0, halfPoint);
	copyArrayOfRange <<<1, 1 >>>(highB, b, halfPoint, sizeB);
	copyArrayOfRange <<<1, 1 >>>(lowB, b, 0, halfPoint);
	hipDeviceSynchronize();


	//printf("%d %d pos 3\n", a[0], b[0]);

	int* lowParts;
	int sizeLowParts = (halfPoint * 2);
	hipMalloc((void**)&lowParts, sizeLowParts * sizeof(int));
	int* highParts;
	int sizeHighParts = (sizeA + sizeB - halfPoint * 2);
	hipMalloc((void**)&highParts, sizeHighParts * sizeof(int));
	int* sumA;
	int sizeSumA = ((halfPoint >= (sizeA - halfPoint)) ? halfPoint : (sizeA - halfPoint));
	hipMalloc((void**)&sumA, sizeSumA * sizeof(int));
	int* sumB;
	int sizeSumB = ((halfPoint >= (sizeB - halfPoint)) ? halfPoint : (sizeB - halfPoint));
	hipMalloc((void**)&sumB, sizeSumB * sizeof(int));

	polyAdd <<< 1, 1 >>> (sumA, lowA, highA, halfPoint, (sizeA - halfPoint));
	polyAdd <<< 1, 1 >>> (sumB, lowB, highB, halfPoint, (sizeB - halfPoint));



	int* sumParts;
	int sizeSumParts = sizeSumA + sizeSumB - 1;
	hipMalloc((void**)&sumParts, sizeSumParts * sizeof(int));

	hipDeviceSynchronize();
	//printf("%d %d pos 4\n", a[0], b[0]);

	

	karatsubaMulKernel <<< 1, 1 >>> (lowParts, lowA, lowB, halfPoint, halfPoint);
	karatsubaMulKernel <<< 1, 1 >>> (sumParts, sumA, sumB, sizeSumA, sizeSumB);
	karatsubaMulKernel <<< 1, 1 >>> (highParts, highA, highB, (sizeA - halfPoint), (sizeB - halfPoint));

	//printf("%d %d pos 5\n", a[0], b[0]);

	/*printf("highB: ");
	for (int i = 0; i < (sizeB - halfPoint); i++)
	{
		printf("%d ", highB[i]);
	}
	printf("\n");*/

	/*printf("highParts: ");
	for (int i = 0; i < (sizeA + sizeB - 2 *halfPoint); i++)
	{
		printf("%d ", highParts[i]);
	}
	printf("\n");
	printf("SUM : %d |", sizeSumA);
	for (int i = 0; i < sizeSumParts; i++)
	{
		printf("%d ", sumParts[i]);
	}
	printf("\n");*/
	hipDeviceSynchronize();

	hipFree(lowA);
	hipFree(lowB);
	hipFree(highA);
	hipFree(highB);
	hipFree(sumA);
	hipFree(sumB);

	int* shiftHighParts;
	int sizeShiftHighParts = (sizeA + sizeB);
	hipMalloc((void**)&shiftHighParts, sizeShiftHighParts * sizeof(int));

	int* sumMinusHigh;
	int sizeHighMinusSum = MAX(sizeHighParts, sizeSumParts);
	hipMalloc((void**)&sumMinusHigh,  sizeHighMinusSum * sizeof(int));

	polyShiftKernel <<< 1, 1 >>> (shiftHighParts, highParts, (sizeA + sizeB - halfPoint * 2), 2 * halfPoint);
	polySubKernel <<< 1, 1 >>> (sumMinusHigh, sumParts, highParts, sizeSumParts, sizeHighParts);

	hipDeviceSynchronize();
	//printf("%d %d pos 6\n", a[0], b[0]);
	/*printf("SUM MINUS HIGH: ");
	for (int i = 0; i < sizeHighMinusSum; i++)
	{
		printf("%d ", sumMinusHigh[i]);
	}
	printf("\n");*/

	int* shiftPlusLow;
	int sizeShiftPlusLow = MAX(sizeShiftHighParts, sizeLowParts);
	hipMalloc((void**)&shiftPlusLow, sizeShiftPlusLow * sizeof(int));

	int* minusMinus;
	int sizeMinusMinus = MAX(sizeHighMinusSum, sizeLowParts);
	hipMalloc((void**)&minusMinus, sizeMinusMinus * sizeof(int));

	polyAdd <<< 1, 1 >>> (shiftPlusLow, shiftHighParts, lowParts, sizeShiftHighParts, sizeLowParts);
	polySubKernel <<< 1, 1 >>> (minusMinus, sumMinusHigh, lowParts, sizeHighMinusSum, sizeLowParts);

	hipDeviceSynchronize();
	//printf("%d %d pos 7\n", a[0], b[0]);

	int* finalShift;
	int sizeFinalShift = sizeMinusMinus + halfPoint;
	hipMalloc((void**)&finalShift, sizeFinalShift * sizeof(int));

	polyShiftKernel <<< 1, 1 >>> (finalShift, minusMinus, sizeMinusMinus, halfPoint);

	hipDeviceSynchronize();
	//printf("%d %d pos 8\n", a[0], b[0]);

	/*printf("FINAL SHIFT: ");
	for (int i = 0; i < sizeFinalShift; i++)
	{
		printf("%d ", finalShift[i]);
	}
	printf("\n");*/

	polyAdd <<< 1, 1 >>> (result, shiftPlusLow, finalShift, sizeShiftPlusLow, sizeFinalShift);

	hipFree(lowParts);
	hipFree(highParts);
	hipFree(sumParts);
	hipFree(shiftHighParts);
	hipFree(sumMinusHigh);
	hipFree(minusMinus);
	hipFree(shiftPlusLow);
	hipFree(finalShift);

	hipDeviceSynchronize();
	//printf("%d %d pos 9\n", a[0], b[0]);

	/*printf("RESULT: ");
	for (int i = 0; i < (sizeA + sizeB - 1); i++)
	{
		printf("%d ", result[i]);
	}
	printf("\n");
	printf("\n");*/

	//printf("EXIT sizes: %d %d\n", sizeA, sizeB);
	//return polyAdd(polyAdd(polyShift(r, 2 * halfPoint), polyShift(polySub(polySub(q, r), p), halfPoint)), p);
}


void generatePoly(int* poly, int size, int maxValue)
{
	srand(time(NULL));
	for (int i = 0; i <= size; i++) 
	{
		poly[i] = rand() % maxValue;
	}
}

int main()
{
	int* a;
	int* b;
	int* resultSimple;
	int* resultKaratsuba;
	const int degreeSize = 9;
	const int degrees[degreeSize] = { 1, 2, 3, 8, 16, 32, 64, 128, 256 };//, 512, 1024};
	clock_t begin;
	clock_t end;
	
	int x[] = { 1, 2, 3, 4 };
	int y[] = { 5, 6, 7, 8 };

	/*resultSimple = (int*)malloc((2 * 4 - 1) * sizeof(int));
	resultKaratsuba = (int*)malloc((2 * 4 - 1) * sizeof(int));

	hipError_t cudaStatus = simpleMulWithCuda(resultSimple, x, y, 4, 4, MIN(100, ((2 * 3 - 1) / 5 + 1)));
	cudaStatus = karatsubaMulWithCuda(resultKaratsuba, x, y, 4, 4);

	printPoly(resultSimple, 2 * 4 - 1);
	printf("\n");
	printPoly(resultKaratsuba, 2 * 4 - 1);
	printf("\n");

	free(resultSimple);
	free(resultKaratsuba);*/

	for (int i = 0; i < degreeSize; i++)
	{
		a = (int*)malloc((degrees[i] + 1) * sizeof(int));
		b = (int*)malloc((degrees[i] + 1) * sizeof(int));
		resultSimple = (int*)malloc((2 * degrees[i] - 1) * sizeof(int));
		resultKaratsuba = (int*)malloc((2 * degrees[i] - 1) * sizeof(int));
		generatePoly(a, degrees[i], 10);
		generatePoly(b, degrees[i], 10);

		printf("degree: %d: ", degrees[i]);
		begin = clock();
		hipError_t cudaStatus = simpleMulWithCuda(resultSimple, a, b, degrees[i], degrees[i], MIN(100, ((2 * degrees[i] - 1) / 5 + 1)));
		end = clock();
		printf("%d ", (int)(double(end - begin) / CLOCKS_PER_SEC * 1000));
		begin = clock();
		cudaStatus = karatsubaMulWithCuda(resultKaratsuba, a, b, degrees[i], degrees[i]); 
		end = clock();
		printf("%d\n", (int)(double(end - begin) / CLOCKS_PER_SEC * 1000));
		/*if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}*/

		/*printPoly(resultSimple, 2 * degrees[i] - 1);
		printf("\n");
		printPoly(resultKaratsuba, 2 * degrees[i] - 1);
		printf("\n");*/


		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
		free(a);
		free(b);
		free(resultSimple);
		free(resultKaratsuba);
	}
	getchar();

    return 0;
}


hipError_t simpleMulWithCuda(int *result, const int *a, const int *b, unsigned int sizeA, unsigned int sizeB, unsigned int numberOfThreads)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_result = 0;

	//int numberOfThreads = sizeA;
	//int sizeB = sizeA;

	int sizeResult = sizeA + sizeB - 1;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_result, sizeResult * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, sizeA * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, sizeB * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, sizeA * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizeB * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    simpleMulKernel<<<1, numberOfThreads>>>(dev_result, dev_a, dev_b, sizeA, sizeB, numberOfThreads);
	//karatsubaMulKernel <<<1, 1 >>>(dev_result, dev_a, dev_b, sizeA, sizeB);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(result, dev_result, sizeResult * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_result);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

hipError_t karatsubaMulWithCuda(int *result, const int *a, const int *b, unsigned int sizeA, unsigned int sizeB)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_result = 0;

	//int numberOfThreads = sizeA;
	//int sizeB = sizeA;

	int sizeResult = sizeA + sizeB - 1;
	hipError_t cudaStatus;


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_result, sizeResult * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, sizeA * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, sizeB * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, sizeA * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, sizeB * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	//simpleMulKernel<<<1, numberOfThreads>>>(dev_result, dev_a, dev_b, sizeA, sizeB, numberOfThreads);
	karatsubaMulKernel <<<1, 1 >>>(dev_result, dev_a, dev_b, sizeA, sizeB);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, sizeResult * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
